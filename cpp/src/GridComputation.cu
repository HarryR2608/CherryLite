#include "hip/hip_runtime.h"
#include "GridComputation.cuh"

#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void ComputeGrid(int* grid, int* memory, bool* direction, int width, int height) {
    memory[0] = grid[0];
    for (int j = 0; j < height; j++) {
        for (int i = 0; i < width; i++) {
            if (j == 0 && i != 0) {
                memory[i] = memory[i-1] + grid[i];
                direction[i] = true;
            }
            else if (i == 0 && j != 0)
            {
                memory[j * width] = memory[(j - 1) * width] + grid[j * width];
                direction[j * width] = false;
            }
            else if (i != 0 && j != 0)
            {
                if (memory[((j - 1) * width) + i] < memory[(j * width) + i - 1]) {
                    memory[(j * width) + i] = memory[((j - 1) * width) + i] + grid[(j * width) + i];
                    direction[(j * width) + i] = false;
                }
                else {
                    memory[(j * width) + i] = memory[(j * width) + i - 1] + grid[(j * width) + i];
                    direction[(j * width) + i] = true;
                }
            }
        }
    }
}

std::string GridComputation::Execute(std::string gridString) {

    //DIRECTION 1 means FROM THE LEFT, 0 means FROM THE TOP

    int width = 1;
    int height = 1;
    for (char element : gridString) {
        if (element == '_') {
            width++;
            height++;
        }
        else if (element == '-') {
            width++;
        }
    }
    width /= height;

    //Host variables
    int* grid;
    int* memory;
    bool* direction;

    //Device variables
    int* d_grid;
    int* d_memory;
    bool* d_direction;

    //Allocate host memory
    grid = (int*)malloc(sizeof(int) * width * height);
    memory = (int*) malloc(sizeof(int) * width * height);
    direction = (bool*) malloc(sizeof(bool) * width * height);

    //Initialise grid
    size_t elementPosition;
    size_t rowPosition;
    std::string elementDelimiter = "-";
    std::string rowDelimiter = "_";
    std::string element;
    std::string row;
    int i = 0;
    while((rowPosition = gridString.find(rowDelimiter)) != std::string::npos) {
        row = gridString.substr(0, rowPosition);

        while ((elementPosition = row.find(elementDelimiter)) != std::string::npos) {
            element = row.substr(0, elementPosition);
            grid[i] = std::stoi(element);
            i++;
            row.erase(0, elementPosition + elementDelimiter.length());
        }
        grid[i] = std::stoi(row);
        i++;
        gridString.erase(0, rowPosition + rowDelimiter.length());
    }
    while ((elementPosition = gridString.find(elementDelimiter)) != std::string::npos) {
        element = gridString.substr(0, elementPosition);
        grid[i] = std::stoi(element);
        i++;
        gridString.erase(0, elementPosition + elementDelimiter.length());
    }
    grid[i] = std::stoi(gridString);

    //Allocate device memory
    hipMalloc((void**)&d_grid, sizeof(int) * width * height);
    hipMalloc((void**)&d_memory, sizeof(int) * width * height);
    hipMalloc((void**)&d_direction, sizeof(bool) * width * height);

    //Copy data to device
    hipMemcpy(d_grid, grid, sizeof(int) * width * height, hipMemcpyHostToDevice);

    //Execute kernel
    ComputeGrid<<<1,1>>>(d_grid, d_memory, d_direction, width, height);

    //Copy data to host
    hipMemcpy(direction, d_direction, sizeof(bool) * width * height, hipMemcpyDeviceToHost);

    //Calculate route
    int currentI = width - 1;
    int currentJ = height - 1;
    std::string route = "";
    for (int a = 0; a < width + height - 2; a++) {
        route = (direction[currentJ * width + currentI] ? "1" : "0") + route;
        if (direction[currentJ * width + currentI]) {
            currentI--;
        } else {
            currentJ--;
        }
    }

    // Deallocate device memory
    hipFree(d_grid);
    hipFree(d_memory);
    hipFree(d_direction);

    // Deallocate host memory
    free(grid);
    free(memory);
    free(direction);

    return route;
}